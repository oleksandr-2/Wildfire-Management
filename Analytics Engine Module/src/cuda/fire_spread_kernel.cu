#include <hip/hip_runtime.h>

#include <cmath>

// Constants for grid dimensions
const int GRID_WIDTH = 1024;
const int GRID_HEIGHT = 1024;

// Define kernel for fire spread simulation
__global__ void fireSpreadKernel(float* grid, float* newGrid, int width, int height, float spreadFactor) 
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) 
    {
        int index = y * width + x;

        // Compute the new fire intensity
        float newIntensity = grid[index];
        if (grid[index] > 0.0f) 
        {
            // Spread to neighboring cells
            if (x > 0) newIntensity += grid[index - 1] * spreadFactor;
            if (x < width - 1) newIntensity += grid[index + 1] * spreadFactor;
            if (y > 0) newIntensity += grid[index - width] * spreadFactor;
            if (y < height - 1) newIntensity += grid[index + width] * spreadFactor;

            newIntensity = fminf(newIntensity, 1.0f); // Cap intensity
        }

        newGrid[index] = newIntensity;
    }
}

// Host function to launch the kernel
void runFireSpreadSimulation(float* h_grid, float* h_newGrid, int width, int height, float spreadFactor) 
{
    float* d_grid;
    float* d_newGrid;

    size_t size = width * height * sizeof(float);

    // Allocate device memory
    hipMalloc(&d_grid, size);
    hipMalloc(&d_newGrid, size);

    // Copy data to device
    hipMemcpy(d_grid, h_grid, size, hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // Launch kernel
    fireSpreadKernel<<<gridSize, blockSize>>>(d_grid, d_newGrid, width, height, spreadFactor);

    // Copy results back to host
    hipMemcpy(h_newGrid, d_newGrid, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_grid);
    hipFree(d_newGrid);
}
