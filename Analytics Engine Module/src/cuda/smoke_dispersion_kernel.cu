#include <hip/hip_runtime.h>

#include <cmath>

// Constants for grid dimensions
const int GRID_WIDTH = 1024;
const int GRID_HEIGHT = 1024;

// Define kernel for smoke dispersion simulation
__global__ void smokeDispersionKernel(float* grid, float* newGrid, int width, int height, float dispersionRate) 
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) 
    {
        int index = y * width + x;

        // Compute the new smoke concentration
        float newConcentration = grid[index];
        if (grid[index] > 0.0f) 
        {
            // Spread to neighboring cells with dispersion effect
            float dispersion = 0.0f;
            if (x > 0) dispersion += grid[index - 1] * dispersionRate;
            if (x < width - 1) dispersion += grid[index + 1] * dispersionRate;
            if (y > 0) dispersion += grid[index - width] * dispersionRate;
            if (y < height - 1) dispersion += grid[index + width] * dispersionRate;

            newConcentration += dispersion;
            newConcentration = fminf(newConcentration, 1.0f); // Cap concentration
        }

        newGrid[index] = newConcentration;
    }
}

// Host function to launch the kernel
void runSmokeDispersionSimulation(float* h_grid, float* h_newGrid, int width, int height, float dispersionRate) 
{
    float* d_grid;
    float* d_newGrid;

    size_t size = width * height * sizeof(float);

    // Allocate device memory
    hipMalloc(&d_grid, size);
    hipMalloc(&d_newGrid, size);

    // Copy data to device
    hipMemcpy(d_grid, h_grid, size, hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // Launch kernel
    smokeDispersionKernel<<<gridSize, blockSize>>>(d_grid, d_newGrid, width, height, dispersionRate);

    // Copy results back to host
    hipMemcpy(h_newGrid, d_newGrid, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_grid);
    hipFree(d_newGrid);
}
